#include "hip/hip_runtime.h"
% Params(3) = 6;
% Params(4) = 50000;
% Params(5) = 25; 

Params(2) = Nfilt;
Params(3) = 5;
Params(4) = 50000;
Params(5) = 50; 

% ParamsW = Params;
% ParamsW(2)= Nrank*Nfilt;
% utu = gpuArray.ones(Nrank*Nfilt, 'single');
% wtw = mexWtW(ParamsW, W(:,:), utu);
% wtw = reshape(wtw, Nfilt, Nrank, Nfilt, Nrank, 2*nt0-1);

U0 = gpuArray(U);
WtW  = gpuArray.zeros(Nfilt,Nfilt, 2*nt0-1, 'single');
for i = 1:Nrank
    for j = 1:Nrank
        utu0 = U0(:,:,i)' * U0(:,:,j);
        wtw0 = mexWtW2(Params, W(:,:,i), W(:,:,j), utu0);
%         wtw0 = squeeze(wtw(:,i,:,j,:));
        WtW = WtW + wtw0;
    end
end
mWtW = max(WtW, [], 3);
mWtW = gather(mWtW);

WtW = permute(WtW, [3 1 2]);
%
rez.WtW = gather(WtW);
clear wtw0 utu0 U0
%
clear nspikes2
st3 = [];

if ops.verbose
   fprintf('Time %3.0fs. Running the final template matching pass...\n', toc) 
end

fid = fopen(fullfile(root, fnameTW), 'r');
msg = [];

% Mask = abs(mWtW-diag(diag(mWtW)))>.05;
Mask = gather(abs(WtW)>.01);
nt1 = nt0;

for ibatch = 1:Nbatch
    if ibatch>Nbatch_buff
        offset = 2 * ops.Nchan*batchstart(ibatch-Nbatch_buff);
        fseek(fid, offset, 'bof');
        dat = fread(fid, [NT ops.Nchan], '*int16');
    else
       dat = DATA(:,:,ibatch); 
    end
    dataRAW = gpuArray(dat);
    dataRAW = single(dataRAW);
    dataRAW = dataRAW / ops.scaleproc;
    data 	= dataRAW * U(:,:); 
    
    [st, id, x] = mexMPmuLITE(Params,data,W(:,:),WtW, mu, lam * 20./mu);
   
    nspikes2(1:size(W,2)+1, ibatch) = histc(id, 0:1:size(W,2));
    
    [~, isort] = sort(st, 'ascend');
    st0 = st(isort);
    x0  = x(isort);
    id0 = 1+ id(isort);
    isiso = get_isolated(st0, id0, Mask, nt1);
  
%     SORT THESE CORRECTLY BEFORE RUNNING AGAIN
    
    inds0 = repmat(double(st0' + (id0'-1)*NT), nt0, 1) + repmat((1:nt0)', 1, numel(st0));
    
    coefs = zeros(numel(st0), Nrank);
    for irank = 1:Nrank
        inds = inds0 + (irank-1)*Nfilt*NT;
        ww = reshape(data(inds), nt0, []);
        coefs(:,irank) = gather(sum(W(:,id0,irank) .* ww, 1));
    end
    
    %%
    STT = cat(2, double(st0) +(NT-ops.ntbuff)*(ibatch-1), double(id0)+1, double(x0), double(isiso), coefs);
    st3 = cat(1, st3, STT);
    
%     keyboard;
    
    if rem(ibatch,100)==1
        nsort = sort(sum(nspikes2,2), 'descend');
        fprintf(repmat('\b', 1, numel(msg)));
        msg = sprintf('Time %2.2f, batch %d/%d, err %2.6f, NTOT %d, n100 %d, n200 %d, n300 %d, n400 %d\n', ...
            toc, ibatch,Nbatch, nanmean(delta), sum(nspikes2(:)), nsort(100), nsort(200), ...
            nsort(min(size(W,2), 300)), nsort(min(size(W,2), 400)));
        fprintf(msg);
    end
end

nsort = sort(sum(nspikes2,2), 'descend');
fprintf('Time %3.0fs. ExpVar %2.6f, n10 %d, n20 %d, n30 %d, n40 %d \n', toc, nanmean(delta), nsort(10), nsort(20), ...
    nsort(min(size(W,2), 30)), nsort(min(size(W,2), 40)));

fclose(fid);
%%
fprintf('Time %3.0fs. Thresholding spikes at false positive rate...\n', toc) 
st3pos = [];
fprate = ops.fprate;
Thx = zeros(Nfilt,1);
for idd = 1:1:Nfilt
    ix = find(st3(:,2)==idd);
    xs = st3(ix, 3);
    
    Mu = 10*ops.Th;
    Nbins = 1000;
    
    bbins = linspace(0, Mu, Nbins);
    hpos = cumsum(hist(Mu - xs(xs>0), bbins));
    hneg = cumsum(hist(Mu + xs(xs<0), bbins));
    
    ifirst = find(hneg./hpos > fprate, 1);
    if isempty(ifirst)
        ifirst = numel(bbins);
    end
    Thx(idd) = Mu - bbins(ifirst);
    
    st3pos = cat(1, st3pos, st3(ix(xs>Thx(idd)), :));
end

[~, isort] = sort(st3pos(:,1), 'ascend');
st3pos = st3pos(isort,:);

rez.st3      = st3; 
rez.st3pos   = st3pos; 
rez.ops      = ops;

% WUnorms = sum(sum(dWUtotCPU.^2, 2), 1).^.5;
% rez.template = gather(dWUtotCPU ./ repmat(WUnorms, nt0, Nchan, 1));

rez.W = W;
rez.U = U;
rez.t2p = [];
for i = 1:Nfilt
    wav0 = W(:,i,1);
    wav0 = my_conv(wav0', .5)';
   [~, itrough] = min(wav0);
    [~, t2p] = max(wav0(itrough:end));
    rez.t2p(i,1) = t2p;
    rez.t2p(i,2) = itrough;   
end

rez.nbins = histc(rez.st3pos(:,2), .5:1:Nfilt+1);

[~, rez.ypos] = max(rez.U(:,:,1), [], 1);

% estimateErrors;
%%
