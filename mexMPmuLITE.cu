#include "hip/hip_runtime.h"
/*
 * Example of how to use the mxGPUArray API in a MEX file.  This example shows
 * how to write a MEX function that takes a gpuArray input and returns a
 * gpuArray output, e.g. B=mexFunction(A).
 *
 * Copyright 2012 The MathWorks, Inc.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdint.h>
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <cstdlib>
#include <algorithm>
#include <iostream>
using namespace std;

const int nt0 = 61,  Nthreads = 1024, lockout = nt0-1, NchanMax = 128, block = 32, NrankMax = 3;
//////////////////////////////////////////////////////////////////////////////////////////
__global__ void	Conv1D(const double *Params, const float *data, const float *W, float *conv_sig){    
  volatile __shared__ float  sW[nt0*NrankMax], sdata[(Nthreads+nt0)*NrankMax]; 
  float x;
  int tid, tid0, bid, i, nid, Nrank, NT, Nfilt;

  tid 		= threadIdx.x;
  bid 		= blockIdx.x;
  Nfilt    	=   (int) Params[1];
  NT      	=   (int) Params[0];
  Nrank     = (int) Params[6];
  
  if(tid<nt0*((int) Params[6]))
      sW[tid]= W[tid%nt0 + (bid + Nfilt * (tid/nt0))* nt0];
  __syncthreads();
  
  tid0 = 0;
  while (tid0<NT-Nthreads-nt0+1){
	  if (tid<nt0*NrankMax) sdata[tid%nt0 + (tid/nt0)*(Nthreads+nt0)] = 
			data[tid0 + tid%nt0+ NT*(bid + Nfilt*(tid/nt0))];
	  #pragma unroll 3
      for(nid=0;nid<Nrank;nid++){
          sdata[tid + nt0+nid*(Nthreads+nt0)] = data[nt0+tid0 + tid+ NT*(bid +nid*Nfilt)];
	  }
	  __syncthreads();
      
	  x = 0.0f;
      for(nid=0;nid<Nrank;nid++){
		  #pragma unroll 4
          for(i=0;i<nt0;i++)
              x    += sW[i + nid*nt0] * sdata[i+tid + nid*(Nthreads+nt0)];
	  }
      conv_sig[tid0  + tid + NT*bid]   = x;
      
      tid0+=Nthreads;
      __syncthreads();
  }
}
//////////////////////////////////////////////////////////////////////////////////////////
__global__ void  bestFilter(const double *Params, const float *data, 
	const float *mu, const float *lam, float *xbest, float *err, int *ftype){
  int tid, tid0, i, bid, NT, Nfilt, ibest = 0;
  float Th,  Cf, Ci, xb, Cbest = 0.0f;

  tid 		= threadIdx.x;
  bid 		= blockIdx.x;
  NT 		= (int) Params[0];
  Nfilt 	= (int) Params[1];
  Th 		= (float) Params[2];
 
  tid0 = tid + bid * Nthreads;
  if (tid0<NT){
    for (i=0; i<Nfilt;i++){
      Ci = data[tid0 + NT * i] + mu[i] * lam[i];
      Cf = Ci * Ci / (lam[i] + 1.0f) - lam[i]*mu[i]*mu[i];
		if (Cf > Cbest){
			Cbest 	= Cf;
			xb      = Ci - lam[i] * mu[i]; // /(lam[i] + 1);
			ibest 	= i;
		}
    }
    if (Cbest > Th*Th){
      err[tid0] 	= Cbest;
      xbest[tid0] 	= xb;
      ftype[tid0] 	= ibest;
    }
  }
}
//////////////////////////////////////////////////////////////////////////////////////////
__global__ void	cleanup_spikes(const double *Params, const float *xbest, const float *err, 
	const int *ftype, int *st, int *id, float *x, float *C, int *counter){
  int indx, maxFR, NTOT, tid, bid, NT, tid0,  j;
  volatile __shared__ float sdata[Nthreads+2*lockout+1];
  bool flag=0;
  float err0;
  
  tid 		= threadIdx.x;
  bid 		= blockIdx.x;
  
  NT      	=   (int) Params[0];
  maxFR 	= (int) Params[3];
  tid0 		= bid * Nthreads;


  if(tid0<NT-Nthreads-2*lockout-1){       
    if (tid<2*lockout)
      sdata[tid] = err[tid0 + tid];
    sdata[tid+2*lockout] = err[2*lockout + tid0 + tid];

    __syncthreads();
    
    err0 = sdata[tid+lockout];
    if(err0>1e-10){
        flag = 0;
        for(j=-lockout;j<=lockout;j++)
            if(sdata[tid+lockout+j]>err0){
                flag = 1;
                break;
            }
        if(flag==0){
            indx = atomicAdd(&counter[0], 1);
            if (indx<maxFR){
                st[indx] = tid+lockout         + tid0;
                id[indx] = ftype[tid+lockout   + tid0];
                x[indx]  = xbest[tid+lockout     + tid0];
                C[indx]  = err0;
            }
        }
    }
  }
}
//////////////////////////////////////////////////////////////////////////////////////////
__global__ void	subSpikes(const double *Params, const int *st, const int *id, const float *x, const int *counter, float *dout, const float *WtW){
  int tid, bid,  NT, ind, tcurr, Nfilt;
  tid 		= threadIdx.x;
  bid 		= blockIdx.x;
  NT 		= (int) Params[0];
  Nfilt 	= (int) Params[1];

  for(ind=counter[1]; ind<counter[0];ind++){
    tcurr = tid + st[ind]-nt0+1;
    if (tcurr>=0 & tcurr<NT)
      dout[tcurr + bid*NT] -= x[ind] * WtW[tid + id[ind]*(2*nt0-1) + (2*nt0-1)*Nfilt*bid];
  }
}
//////////////////////////////////////////////////////////////////////////////////////////
__global__ void	subtract_spikes(const double *Params,  const int *st, const int *id, const float *x, const int *counter, float *dataraw, const float *W, const float *U){
  int tid, bid, Nblocks, i, NT, ind, Nchan;
  __shared__ float sh_W[nt0], sh_U[NchanMax];
  tid 		= threadIdx.x;
  bid 		= blockIdx.x;
  Nblocks   = gridDim.x;
  NT        = (int) Params[0];
  Nchan     = (int) Params[5];
  ind       = bid;

  while(ind<counter[0]){
    if (tid<nt0) sh_W[tid] = W[tid + nt0*id[ind]];
    sh_U[tid] = U[tid + Nchan*id[ind]];

    __syncthreads();
    for (i=0;i<nt0;i++)
      dataraw[i + st[ind] + NT * tid] -= x[ind] * sh_W[i] * sh_U[tid];
    ind+= Nblocks;
    __syncthreads();
  }

}
//////////////////////////////////////////////////////////////////////////////////////////
__global__ void getWgradient(const double *Params, const int *st, const int *id, const float *x,  const int *counter, const float *datarez, const float *U, float *dW){
  int tid, bid, i, ind, NT, Nchan;
  float xprod; 
  volatile __shared__ float sh_U[NchanMax];
  NT = (int) Params[0];
    Nchan = (int) Params[5];

  tid 		= threadIdx.x;
  bid 		= blockIdx.x;
  while(tid<Nchan){
    sh_U[tid] = U[tid + bid*Nchan];
    tid+= blockDim.x;
  }
  tid 		= threadIdx.x;
  __syncthreads();

  for(ind=0; ind<counter[0];ind++)
    if (id[ind]==bid){
      xprod = 0.0f;
      for (i=0;i<Nchan;i++)
	xprod+= sh_U[i] * datarez[st[ind] + tid + NT * i];
      dW[tid + nt0 * bid] += xprod * x[ind];
    }
}
//////////////////////////////////////////////////////////////////////////////////////////
__global__ void getUgradient(const double *Params, const int *st, const int *id, const float *x,  const int *counter, const float *datarez, const float *W, float *dU){  
  int j, tid, bid, i, ind, NT, Nchan;
  float xprod; 
  volatile __shared__ float sh_M[NchanMax*nt0], sh_W[nt0];

  NT = (int) Params[0];
    Nchan = (int) Params[5];

  tid 		= threadIdx.x;
  bid 		= blockIdx.x;
  if (tid<nt0)
    sh_W[tid] = W[tid + nt0*bid];
 
  __syncthreads();

  for(ind=0; ind<counter[0];ind++)
    if (id[ind]==bid){
      if(tid<nt0)
	for (j=0;j<Nchan;j++)
	  sh_M[tid + nt0*j] = datarez[tid + st[ind] + NT*j];
      __syncthreads();

      xprod = 0.0f;
      for (i=0;i<nt0;i++)
	xprod+= sh_W[i] * sh_M[i + tid*nt0];
      dU[tid + bid*Nchan] += xprod * x[ind];
      __syncthreads();
    }  
}
//////////////////////////////////////////////////////////////////////////////////////////

/*
 * Host code
 */
void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
  /* Declare input variables*/
  double *Params, *d_Params;
  int blocksPerGrid, NT, maxFR, Nchan;
  int const threadsPerBlock = Nthreads;

  /* Initialize the MathWorks GPU API. */
  mxInitGPU();

  /* read Params and copy to GPU */
  Params  	= (double*) mxGetData(prhs[0]);
  NT		= (int) Params[0];
  blocksPerGrid	= (int) Params[1];
  maxFR		= (int) Params[3];
   Nchan = (int) Params[5];

  hipMalloc(&d_Params,      sizeof(double)*mxGetNumberOfElements(prhs[0]));
  hipMemcpy(d_Params,Params,sizeof(double)*mxGetNumberOfElements(prhs[0]),hipMemcpyHostToDevice);

  /* collect input GPU variables*/
  mxGPUArray const  *W,   *data,   *WtW, *mu,   *lam;
  const float     *d_W, *d_data, *d_WtW,  *d_mu, *d_lam;
  
  data       = mxGPUCreateFromMxArray(prhs[1]);
  d_data     = (float const *)(mxGPUGetDataReadOnly(data));
  W             = mxGPUCreateFromMxArray(prhs[2]);
  d_W        	= (float const *)(mxGPUGetDataReadOnly(W));
  WtW       	= mxGPUCreateFromMxArray(prhs[3]);
  d_WtW     	= (float const *)(mxGPUGetDataReadOnly(WtW));
  mu            = mxGPUCreateFromMxArray(prhs[4]);
  d_mu          = (float const *)(mxGPUGetDataReadOnly(mu));
  lam       	= mxGPUCreateFromMxArray(prhs[5]);
  d_lam     	= (float const *)(mxGPUGetDataReadOnly(lam));

  /* allocate new GPU variables*/  
  float *d_err,*d_C, *d_xbest, *d_x, *d_dout;
  int *d_st,  *d_ftype,  *d_id, *d_counter;

  hipMalloc(&d_dout,   NT * blocksPerGrid* sizeof(float));

  hipMalloc(&d_err,   NT * sizeof(float));
  hipMalloc(&d_xbest,   NT * sizeof(float));
  hipMalloc(&d_ftype, NT * sizeof(int));
  hipMalloc(&d_st,    maxFR * sizeof(int));
  hipMalloc(&d_id,    maxFR * sizeof(int));
  hipMalloc(&d_x,     maxFR * sizeof(float));
  hipMalloc(&d_C,     maxFR * sizeof(float));
  hipMalloc(&d_counter,   2*sizeof(int));
 
  hipMemset(d_dout,    0, NT * blocksPerGrid * sizeof(float));
  hipMemset(d_counter, 0, 2*sizeof(int));
  hipMemset(d_st,      0, maxFR *   sizeof(int));
  hipMemset(d_id,      0, maxFR *   sizeof(int));
  hipMemset(d_x,       0, maxFR *    sizeof(float));
  hipMemset(d_C,       0, maxFR *    sizeof(float));

  int *counter;
  counter = (int*) calloc(1,sizeof(int));
 
  Conv1D<<<blocksPerGrid,threadsPerBlock>>>(d_Params, d_data, d_W, d_dout); 
  for(int k=0;k<(int) Params[4];k++){
    hipMemset(d_err,     0, NT * sizeof(float));
    hipMemset(d_ftype,   0, NT * sizeof(int));
    hipMemset(d_xbest,   0, NT * sizeof(float));

    bestFilter<<<NT/Nthreads,threadsPerBlock>>>(    d_Params, d_dout, d_mu, d_lam, d_xbest, d_err, d_ftype);
    cleanup_spikes<<<NT/Nthreads,threadsPerBlock>>>(d_Params, d_xbest, d_err, d_ftype, d_st, d_id, d_x, d_C, d_counter);
 
    hipMemcpy(counter, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    if (counter[0]>maxFR){
      counter[0] = maxFR;
      hipMemcpy(d_counter, counter, sizeof(int), hipMemcpyHostToDevice);      
    }
    
    subSpikes<<<blocksPerGrid, 2*nt0-1>>>(d_Params, d_st, d_id, d_x, d_counter, d_dout,    d_WtW);

    hipMemcpy(d_counter+1, d_counter, sizeof(int), hipMemcpyDeviceToHost);

    if(counter[0]==maxFR)
      break;
  }

  float *x, *C;
  int *st, *id;
  int minSize;
  if (counter[0]<maxFR)  minSize = counter[0];
  else                   minSize = maxFR;
  const mwSize dimst[] 	= {minSize,1}; 
  plhs[0] = mxCreateNumericArray(2, dimst, mxINT32_CLASS, mxREAL);
  st = (int*) mxGetData(plhs[0]);
  plhs[1] = mxCreateNumericArray(2, dimst, mxINT32_CLASS, mxREAL);
  id = (int*) mxGetData(plhs[1]);
  plhs[2] = mxCreateNumericArray(2, dimst, mxSINGLE_CLASS, mxREAL);
  x =  (float*) mxGetData(plhs[2]);
  plhs[3] = mxCreateNumericArray(2, dimst, mxSINGLE_CLASS, mxREAL);
  C =  (float*) mxGetData(plhs[3]);
  hipMemcpy(st, d_st, minSize * sizeof(int),   hipMemcpyDeviceToHost);
  hipMemcpy(id, d_id, minSize * sizeof(int),   hipMemcpyDeviceToHost);
  hipMemcpy(x,   d_x, minSize * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(C,   d_C, minSize * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_ftype);
  hipFree(d_err);
  hipFree(d_xbest);
  hipFree(d_st);
  hipFree(d_id);
  hipFree(d_x);
  hipFree(d_C);
  hipFree(d_counter);
  hipFree(d_Params);

  hipFree(d_dout);

  mxGPUDestroyGPUArray(data);
  mxGPUDestroyGPUArray(WtW);
  mxGPUDestroyGPUArray(W);
  mxGPUDestroyGPUArray(mu);
  mxGPUDestroyGPUArray(lam);
}
