#include "hip/hip_runtime.h"
/*
 * Example of how to use the mxGPUArray API in a MEX file.  This example shows
 * how to write a MEX function that takes a gpuArray input and returns a
 * gpuArray output, e.g. B=mexFunction(A).
 *
 * Copyright 2012 The MathWorks, Inc.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdint.h>
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <cstdlib>
#include <algorithm>
#include <iostream>
using namespace std;

const int nt0 = 61,  Nthreads = 1024,   lockout = nt0-1, NchanMax = 128, block = 32;

//////////////////////////////////////////////////////////////////////////////////////////
__global__ void	Conv1D(const double *Params, const float *data, const float *W, float *conv_sig){    
  __shared__ float sW[nt0], sdata[Nthreads+nt0]; 
  float x;
  int tid, tid0, bid, i, NT;

  tid 		= threadIdx.x;
  bid 		= blockIdx.x;
  
  if(tid<nt0)        sW[tid]= W[tid + bid * nt0];
  __syncthreads();
	 	 
  NT      	=   (int) Params[0];
  tid0 = 0;
  while (tid0<NT-Nthreads-nt0+1){    
    if (tid<nt0) sdata[tid] = data[tid0 + tid+ NT*bid];
    sdata[nt0+tid] = data[nt0+tid0 + tid+ NT*bid];
     __syncthreads();
    
    x = 0.0f;
    for(i=0;i<nt0;i++)
      x    += sW[i] * sdata[i+tid];
    
    conv_sig[tid0  + tid + NT*bid]   = x;
    
    tid0+=Nthreads;
     __syncthreads();
  }
}
//////////////////////////////////////////////////////////////////////////////////////////
__global__ void  bestFilter(const double *Params, const float *data, float *err, int *ftype){

  int tid, tid0, i, bid, NT, Nfilt, ibest = 0;
  float xbest = 0.0f, Th;

  tid 		= threadIdx.x;
  bid 		= blockIdx.x;
  NT 		= (int) Params[0];
  Nfilt 	= (int) Params[1];
  Th 		= (float) Params[2];

  tid0 = tid + bid * Nthreads;
  if (tid0<NT){
    for (i=0; i<Nfilt;i++)
      if (abs(data[tid0 + NT * i]) > abs(xbest)){
	xbest = data[tid0 + NT * i];
	ibest = i;
      }
    if (abs(xbest)>Th){
      err[tid0] 	= xbest;
      ftype[tid0] 	= ibest;
    }
  }
}

//////////////////////////////////////////////////////////////////////////////////////////
__global__ void	cleanup_spikes(const double *Params, const float *err, const int *ftype, int *st, int *id, float *x, int *counter){
  int indx, maxFR, NTOT, tid, bid, NT, tid0,  j;
  volatile __shared__ float sdata[Nthreads+2*lockout+1];
  bool flag=0;
  float err0;
  
  tid 		= threadIdx.x;
  bid 		= blockIdx.x;
  
  NT      	=   (int) Params[0];
  maxFR 	= (int) Params[3];
  tid0 		= bid * Nthreads;


  if(tid0<NT-Nthreads-2*lockout-1){       
    if (tid<2*lockout)
      sdata[tid] = abs(err[tid0 + tid]*err[tid0 + tid]);
    sdata[tid+2*lockout] = abs(err[2*lockout + tid0 + tid]*err[2*lockout + tid0 + tid]);

    __syncthreads();
    
    err0 = sdata[tid+lockout];
    if(err0>1e-10){
      flag = 0;
      for(j=-lockout;j<=lockout;j++)
	if(sdata[tid+lockout+j]>err0){
	  flag = 1;
	  break;
	}     
      if(flag==0){
	  indx = atomicAdd(&counter[0], 1);
	  if (indx<maxFR){
	    st[indx] = tid+lockout         + tid0;
	    id[indx] = ftype[tid+lockout   + tid0];
	    x[indx]  = err[tid+lockout     + tid0];
	  }
      }
    }
  }
}
//////////////////////////////////////////////////////////////////////////////////////////
__global__ void	subSpikes(const double *Params, const int *st, const int *id, const float *x, const int *counter, float *dout, const float *WtW){
  int tid, bid,  NT, ind, tcurr, Nfilt, Nchan;
  tid 		= threadIdx.x;
  bid 		= blockIdx.x;
  NT 		= (int) Params[0];
  Nfilt 	= (int) Params[1];
  Nchan         = (int) Params[5];

  for(ind=counter[1]; ind<counter[0];ind++){
    tcurr = tid + st[ind]-nt0+1;
    if (tcurr>=0 & tcurr<NT)
      dout[tcurr + bid*NT] -= x[ind] * WtW[tid + id[ind]*(2*nt0-1) + (2*nt0-1)*Nfilt*bid];
  }
}
//////////////////////////////////////////////////////////////////////////////////////////
__global__ void	subtract_spikes(const double *Params,  const int *st, const int *id, const float *x, const int *counter, float *dataraw, const float *W, const float *U){
  int tid, bid, Nblocks, i, NT, ind, Nchan;
  __shared__ float sh_W[nt0], sh_U[NchanMax];
  tid 		= threadIdx.x;
  bid 		= blockIdx.x;
  Nblocks       = gridDim.x;
  NT = (int) Params[0];
  Nchan         = (int) Params[5];
  ind = bid + counter[1];

  while(ind<counter[0]){
    while (tid<nt0){ sh_W[tid] = W[tid + nt0*id[ind]]; tid+=blockDim.x;}    
    tid 		= threadIdx.x;
    sh_U[tid] = U[tid + Nchan*id[ind]];
    
    __syncthreads();
    for (i=0;i<nt0;i++)
      dataraw[i + st[ind] + NT * tid] -= x[ind] * sh_W[i] * sh_U[tid];
    ind+= Nblocks;
    __syncthreads();
  }

}
//////////////////////////////////////////////////////////////////////////////////////////
__global__ void getWgradient(const double *Params, const int *st, const int *id, 
        const float *x,  const int *counter, const float *datarez, const float *U, float *dW){
  int tid, bid, i, ind, NT, Nchan;
  float xprod; 
  volatile __shared__ float sh_U[NchanMax];
  NT = (int) Params[0];
    Nchan         = (int) Params[5];

  tid 		= threadIdx.x;
  bid 		= blockIdx.x;
  while(tid<Nchan){
    sh_U[tid] = U[tid + bid*Nchan];
    tid+= blockDim.x;
  }
  tid 		= threadIdx.x;
  __syncthreads();
  
  for(ind=0; ind<counter[0];ind++)
      if (id[ind]==bid){
          xprod = 0.0f;
          for (i=0;i<Nchan;i++)
              xprod+= sh_U[i] * datarez[st[ind] + tid + NT * i];
          dW[tid + nt0 * bid] += xprod * x[ind];
      }
}
//////////////////////////////////////////////////////////////////////////////////////////
__global__ void getUgradient(const double *Params, const int *st, const int *id, const float *x,  const int *counter, const float *datarez, const float *W, float *dU){
  
  int j, tid, bid, i, ind, NT, Nchan;
  float xprod; 
  volatile __shared__ float sh_M[NchanMax*nt0], sh_W[nt0];

  NT = (int) Params[0];
  Nchan         = (int) Params[5];

  tid 		= threadIdx.x;
  bid 		= blockIdx.x;
  while(tid<nt0){
    sh_W[tid] = W[tid + nt0*bid];
    tid+=blockDim.x;
  }
  tid 		= threadIdx.x;
 
  __syncthreads();
  
  for(ind=0; ind<counter[0];ind++)
      if (id[ind]==bid){
          while(tid<nt0){
              for (j=0;j<Nchan;j++)
                  sh_M[tid + nt0*j] = datarez[tid + st[ind] + NT*j];
              tid+=blockDim.x;
          }
          tid 		= threadIdx.x;
          __syncthreads();

          xprod = 0.0f;
          for (i=0;i<nt0;i++)
              xprod+= sh_W[i] * sh_M[i + tid*nt0];
          dU[tid + bid*Nchan] += xprod * x[ind];
          __syncthreads();
    }  
}
//////////////////////////////////////////////////////////////////////////////////////////

/*
 * Host code
 */
void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
    /* Declare input variables*/
  double *Params, *d_Params;
  int blocksPerGrid, NT, maxFR, Nchan;
  int const threadsPerBlock = Nthreads;

  /* Initialize the MathWorks GPU API. */
  mxInitGPU();

  /* read Params and copy to GPU */
  Params        = (double*) mxGetData(prhs[0]);
  NT            = (int) Params[0];
  blocksPerGrid	= (int) Params[1];
  maxFR         = (int) Params[3];
  Nchan         = (int) Params[5];
  hipMalloc(&d_Params,      sizeof(double)*mxGetNumberOfElements(prhs[0]));
  hipMemcpy(d_Params,Params,sizeof(double)*mxGetNumberOfElements(prhs[0]),hipMemcpyHostToDevice);
  
  /* collect input GPU variables*/
  mxGPUArray const  *W,   *U,   *dataraw,   *data,   *WtW;
  const float     *d_W, *d_U, *d_dataraw, *d_data, *d_WtW;
  
  dataraw       = mxGPUCreateFromMxArray(prhs[1]);
  d_dataraw     = (float const *)(mxGPUGetDataReadOnly(dataraw));
  W             = mxGPUCreateFromMxArray(prhs[2]);
  d_W        	= (float const *)(mxGPUGetDataReadOnly(W));
  U         	= mxGPUCreateFromMxArray(prhs[3]);
  d_U        	= (float const *)(mxGPUGetDataReadOnly(U));
  data        	= mxGPUCreateFromMxArray(prhs[4]);
  d_data        = (float const *)(mxGPUGetDataReadOnly(data));
  WtW       	= mxGPUCreateFromMxArray(prhs[5]);
  d_WtW     	= (float const *)(mxGPUGetDataReadOnly(WtW));
  
  /* allocate new GPU variables*/
  float *d_err, *d_x, *d_dout;
  int *d_st, *d_ftype,  *d_id, *d_counter;
  
  hipMalloc(&d_dout,   NT * blocksPerGrid* sizeof(float));

  hipMalloc(&d_err,   NT * sizeof(float));
  hipMalloc(&d_ftype, NT * sizeof(int));
  hipMalloc(&d_st,    maxFR * sizeof(int));
  hipMalloc(&d_id,    maxFR * sizeof(int));
  hipMalloc(&d_x,     maxFR * sizeof(float));
  hipMalloc(&d_counter,   2*sizeof(int));
 
  hipMemset(d_dout,    0, NT * blocksPerGrid * sizeof(float));
  hipMemset(d_counter, 0, 2*sizeof(int));
  hipMemset(d_st,      0, maxFR *   sizeof(int));
  hipMemset(d_id,      0, maxFR *   sizeof(int));
  hipMemset(d_x,       0, maxFR *    sizeof(float));


  mxGPUArray *datarez, *dW, *dU;
  float *d_datarez, *d_dW, *d_dU;
  const mwSize dimsu[] 	= {NT,Nchan}; 
  datarez 		= mxGPUCreateGPUArray(2, dimsu, mxSINGLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);  
  d_datarez 		= (float *)(mxGPUGetData(datarez));
  hipMemcpy(d_datarez, d_dataraw,  NT * Nchan * sizeof(float), hipMemcpyDeviceToDevice);

  const mwSize dimsdW[] = {nt0,blocksPerGrid}; 
  dW 		= mxGPUCreateGPUArray(2, dimsdW, mxSINGLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);  
  d_dW 		= (float *)(mxGPUGetData(dW));
  hipMemset(d_dW, 0,  nt0*blocksPerGrid * sizeof(float));

  const mwSize dimsdU[] = {Nchan,blocksPerGrid}; 
  dU 		= mxGPUCreateGPUArray(2, dimsdU, mxSINGLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);  
  d_dU 		= (float *)(mxGPUGetData(dU));
  hipMemset(d_dU, 0,  Nchan*blocksPerGrid * sizeof(float));

  int *counter;
  counter = (int*) calloc(1,sizeof(int));
 
  Conv1D<<<blocksPerGrid,threadsPerBlock>>>(d_Params, d_data, d_W, d_dout); 
  for(int k=0;k<(int) Params[4];k++){
    hipMemset(d_err,     0, NT * sizeof(float));
    hipMemset(d_ftype,   0, NT * sizeof(int));

    bestFilter<<<NT/Nthreads,threadsPerBlock>>>(    d_Params, d_dout, d_err, d_ftype);
    cleanup_spikes<<<NT/Nthreads,threadsPerBlock>>>(d_Params, d_err, d_ftype, d_st, d_id, d_x, d_counter);
 
    hipMemcpy(counter, d_counter, sizeof(int), hipMemcpyDeviceToHost);
    if (counter[0]>maxFR){
      counter[0] = maxFR;
      hipMemcpy(d_counter, counter, sizeof(int), hipMemcpyHostToDevice);      
    }
      
    subtract_spikes<<<128,Nchan>>>(       d_Params, d_st, d_id, d_x, d_counter, d_datarez, d_W, d_U);
    subSpikes<<<blocksPerGrid, 2*nt0-1>>>(d_Params, d_st, d_id, d_x, d_counter, d_dout,    d_WtW);

    hipMemcpy(d_counter+1, d_counter, sizeof(int), hipMemcpyDeviceToHost);

    if(counter[0]==maxFR)
      break;
  }

  getWgradient<<<blocksPerGrid,nt0>>>(  d_Params, d_st, d_id, d_x, d_counter, d_datarez, d_U, d_dW);
  getUgradient<<<blocksPerGrid,Nchan>>>(d_Params, d_st, d_id, d_x, d_counter, d_datarez, d_W, d_dU);

  plhs[0] 	= mxGPUCreateMxArrayOnGPU(datarez);
  plhs[1] 	= mxGPUCreateMxArrayOnGPU(dW);
  plhs[2] 	= mxGPUCreateMxArrayOnGPU(dU);


  float *x;
  int *st, *id;
  int minSize;
  if (counter[0]<maxFR)  minSize = counter[0];
  else                   minSize = maxFR;
  const mwSize dimst[] 	= {minSize,1}; 
  plhs[3] = mxCreateNumericArray(2, dimst, mxINT32_CLASS, mxREAL);
  st = (int*) mxGetData(plhs[3]);
  plhs[4] = mxCreateNumericArray(2, dimst, mxINT32_CLASS, mxREAL);
  id = (int*) mxGetData(plhs[4]);
  plhs[5] = mxCreateNumericArray(2, dimst, mxSINGLE_CLASS, mxREAL);
  x =  (float*) mxGetData(plhs[5]);
  hipMemcpy(st, d_st, minSize * sizeof(int),   hipMemcpyDeviceToHost);
  hipMemcpy(id, d_id, minSize * sizeof(int),   hipMemcpyDeviceToHost);
  hipMemcpy(x,   d_x, minSize * sizeof(float), hipMemcpyDeviceToHost);


  hipFree(d_ftype);
  hipFree(d_err);
  hipFree(d_st);
  hipFree(d_id);
  hipFree(d_x);
  hipFree(d_counter);
  hipFree(d_Params);

  hipFree(d_dout);

  mxGPUDestroyGPUArray(data);
  mxGPUDestroyGPUArray(dataraw);
  mxGPUDestroyGPUArray(WtW);
  mxGPUDestroyGPUArray(datarez);
  mxGPUDestroyGPUArray(W);
  mxGPUDestroyGPUArray(U);
  mxGPUDestroyGPUArray(dW);
  mxGPUDestroyGPUArray(dU);
  
}
