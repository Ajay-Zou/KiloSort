#include "hip/hip_runtime.h"
/*
 * Example of how to use the mxGPUArray API in a MEX file.  This example shows
 * how to write a MEX function that takes a gpuArray input and returns a
 * gpuArray output, e.g. B=mexFunction(A).
 *
 * Copyright 2012 The MathWorks, Inc.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdint.h>
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <cstdlib>
#include <algorithm>
#include <iostream>
using namespace std;

const int nt0 = 61,  Nthreads = 1024,   lockout = nt0-1, NchanMax = 128;
//////////////////////////////////////////////////////////////////////////////////////////
__global__ void	Conv1D(const double *Params, const float *data, const float *W, float *conv_sig){    
  __shared__ float sW[nt0], sdata[Nthreads+nt0]; 
  float x;
  int tid, tid0, bid, i, NT;

  tid 		= threadIdx.x;
  bid 		= blockIdx.x;
  
  if(tid<nt0)        sW[tid]= W[tid + bid * nt0];
  __syncthreads();
	 	 
  NT      	=   (int) Params[0];
  tid0 = 0;
  while (tid0<NT-Nthreads-nt0+1){    
    if (tid<nt0) sdata[tid] = data[tid0 + tid+ NT*bid];
    sdata[nt0+tid] = data[nt0+tid0 + tid+ NT*bid];
     __syncthreads();
    
    x = 0.0f;
    for(i=0;i<nt0;i++)
      x    += sW[i] * sdata[i+tid];
    
    conv_sig[tid0  + tid + NT*bid]   = max(0.0f, x);
    
    tid0+=Nthreads;
     __syncthreads();
  }
}
//////////////////////////////////////////////////////////////////////////////////////////
__global__ void  bestFilter(const double *Params, const float *data, float *err, int *ftype){

  int tid, tid0, i, bid, NT, Nfilt, ibest = 0;
  float xbest = 0.0f, Th;

  tid 		= threadIdx.x;
  bid 		= blockIdx.x;
  NT 		= (int) Params[0];
  Nfilt 	= (int) Params[1];
  Th 		= (float) Params[2];

  tid0 = tid + bid * Nthreads;
  if (tid0<NT){
    for (i=0; i<Nfilt;i++)
      if (abs(data[tid0 + NT * i]) > abs(xbest)){
	xbest = data[tid0 + NT * i];
	ibest = i;
      }
    if (abs(xbest)>Th){
      err[tid0] 	= xbest;
      ftype[tid0] 	= ibest;
    }
  }
}

//////////////////////////////////////////////////////////////////////////////////////////
__global__ void	cleanup_spikes(const double *Params, const float *err, const int *ftype, const bool *UtU, int *st, int *id, float *x, int *counter){
  int curr_token, indx, maxFR, Nfilt, NTOT, tid, bid, NT, tid0,  j;
  volatile __shared__ float sdata[Nthreads+2*lockout+1];
  volatile __shared__ int id_sh[Nthreads+2*lockout+1];
  bool flag=0;
  float err0;
  
  tid 		= threadIdx.x;
  bid 		= blockIdx.x;
  
  NT      	= (int) Params[0];
  Nfilt 	= (int) Params[1];
  maxFR 	= (int) Params[3];
  tid0 		= bid * Nthreads;

  if(tid0<NT-Nthreads-2*lockout-1){       
    if (tid<2*lockout){
		sdata[tid] = abs(err[tid0 + tid]*err[tid0 + tid]);
		id_sh[tid] = ftype[tid0 + tid];
	 }
    sdata[tid+2*lockout] = abs(err[2*lockout + tid0 + tid]*err[2*lockout + tid0 + tid]);
	id_sh[tid+2*lockout] = ftype[2*lockout + tid0 + tid];
	
    __syncthreads();
    
    err0 = sdata[tid+lockout];
	curr_token = id_sh[tid+lockout];
    if(err0>1e-10){
      flag = 0;
      for(j=-lockout;j<=lockout;j++)
	if(sdata[tid+lockout+j]>err0)
		if (UtU[curr_token*Nfilt + id_sh[tid+lockout+j]]){
			flag = 1;
			break;
			}     
      if(flag==0){
	  indx = atomicAdd(&counter[0], 1);
	  if (indx<maxFR){
	    st[indx] = tid+lockout         + tid0;
	    id[indx] = ftype[tid+lockout   + tid0];
	    x[indx]  = err[tid+lockout     + tid0];
	  }
      }
    }
  }
}
//////////////////////////////////////////////////////////////////////////////////////////
__global__ void average_snips(const double *Params, const int *st, const int *id, 
        const float *x,  const int *counter, const float *dataraw, float *WU){
  int tidx, tidy, bid, i, ind, NT, Nchan;
  float xsum = 0.0f; 
  Nchan = (int) Params[5];

  NT = (int) Params[0];
  tidx 		= threadIdx.x;
  tidy 		= threadIdx.y;
  bid 		= blockIdx.x;
  
  for(ind=0; ind<counter[0];ind++)
      if (id[ind]==bid){
		  tidy 		= threadIdx.y;
		  while (tidy<Nchan){	
			xsum = x[ind] * dataraw[st[ind]+tidx + NT * tidy];
			WU[tidx+tidy*nt0 + nt0*Nchan * bid] += xsum;
			tidy+=blockDim.y;
		  }
	  }
}
//////////////////////////////////////////////////////////////////////////////////////////

/*
 * Host code
 */
void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
    /* Declare input variables*/
  double *Params, *d_Params;
  int blocksPerGrid, NT, maxFR, Nchan;
  int const threadsPerBlock = Nthreads;

  /* Initialize the MathWorks GPU API. */
  mxInitGPU();

  /* read Params and copy to GPU */
  Params        = (double*) mxGetData(prhs[0]);
  NT            = (int) Params[0];
  blocksPerGrid	= (int) Params[1];
  maxFR         = (int) Params[3];
  Nchan         = (int) Params[5];
  hipMalloc(&d_Params,      sizeof(double)*mxGetNumberOfElements(prhs[0]));
  hipMemcpy(d_Params,Params,sizeof(double)*mxGetNumberOfElements(prhs[0]),hipMemcpyHostToDevice);
  
  /* collect input GPU variables*/
  mxGPUArray const  *W, *dataraw,   *data, *UtU;
  const float      *d_W, *d_dataraw, *d_data;
  const bool *d_UtU;
  
  dataraw       = mxGPUCreateFromMxArray(prhs[1]);
  d_dataraw     = (float const *)(mxGPUGetDataReadOnly(dataraw));
  W             = mxGPUCreateFromMxArray(prhs[2]);
  d_W        	= (float const *)(mxGPUGetDataReadOnly(W));
  data        	= mxGPUCreateFromMxArray(prhs[3]);
  d_data        = (float const *)(mxGPUGetDataReadOnly(data));
  UtU       	= mxGPUCreateFromMxArray(prhs[4]);
  d_UtU     	= (bool const *)(mxGPUGetDataReadOnly(UtU));
  
  /* allocate new GPU variables*/
  float *d_err, *d_x, *d_dout;
  int *d_st, *d_ftype,  *d_id, *d_counter;
  
  hipMalloc(&d_dout,   NT * blocksPerGrid* sizeof(float));

  hipMalloc(&d_err,   NT * sizeof(float));
  hipMalloc(&d_ftype, NT * sizeof(int));
  hipMalloc(&d_st,    maxFR * sizeof(int));
  hipMalloc(&d_id,    maxFR * sizeof(int));
  hipMalloc(&d_x,     maxFR * sizeof(float));
  hipMalloc(&d_counter,   2*sizeof(int));
 
  hipMemset(d_dout,    0, NT * blocksPerGrid * sizeof(float));
  hipMemset(d_counter, 0, 2*sizeof(int));
  hipMemset(d_st,      0, maxFR *   sizeof(int));
  hipMemset(d_id,      0, maxFR *   sizeof(int));
  hipMemset(d_x,       0, maxFR *    sizeof(float));

  mxGPUArray *dWU;
  float *d_dWU;

  const mwSize dimsdWU[] = {nt0,Nchan,blocksPerGrid}; 
  dWU 		= mxGPUCreateGPUArray(3, dimsdWU, mxSINGLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);  
  d_dWU     = (float *)(mxGPUGetData(dWU));
  hipMemset(d_dWU, 0,  nt0*Nchan*blocksPerGrid * sizeof(float));

  int *counter;
  counter = (int*) calloc(1,sizeof(int));
  hipMemset(d_err,     0, NT * sizeof(float));
  hipMemset(d_ftype,   0, NT * sizeof(int));
  
  Conv1D<<<blocksPerGrid,threadsPerBlock>>>(d_Params, d_data, d_W, d_dout);
  bestFilter<<<NT/Nthreads,threadsPerBlock>>>(    d_Params, d_dout, d_err, d_ftype);
  cleanup_spikes<<<NT/Nthreads,threadsPerBlock>>>(d_Params, d_err, d_ftype, d_UtU, d_st, d_id, d_x, d_counter);
  
  dim3 block(nt0, 1024/nt0);
  average_snips<<<blocksPerGrid,block>>>(  d_Params, d_st, d_id, d_x, d_counter, d_dataraw, d_dWU);

  hipMemcpy(counter, d_counter, sizeof(int), hipMemcpyDeviceToHost);
 
  plhs[0] 	= mxGPUCreateMxArrayOnGPU(dWU);

  
  float    *x;
  int *st, *id;
  int minSize;
  if (counter[0]<maxFR)  minSize = counter[0];
  else                   minSize = maxFR;
  const mwSize dimst[] 	= {minSize,1}; 
  plhs[1] = mxCreateNumericArray(2, dimst, mxINT32_CLASS, mxREAL);
  st = (int*) mxGetData(plhs[1]);
  plhs[2] = mxCreateNumericArray(2, dimst, mxINT32_CLASS, mxREAL);
  id = (int*) mxGetData(plhs[2]);
  plhs[3] = mxCreateNumericArray(2, dimst, mxSINGLE_CLASS, mxREAL);
  x =  (float*) mxGetData(plhs[3]);
  hipMemcpy(st, d_st, minSize * sizeof(int),   hipMemcpyDeviceToHost);
  hipMemcpy(id, d_id, minSize * sizeof(int),   hipMemcpyDeviceToHost);
  hipMemcpy(x,   d_x, minSize * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_ftype);
  hipFree(d_err);
  hipFree(d_st);
  hipFree(d_id);
  hipFree(d_x);
  hipFree(d_counter);
  hipFree(d_Params);

  hipFree(d_dout);

  mxGPUDestroyGPUArray(data);
  mxGPUDestroyGPUArray(dataraw);
  mxGPUDestroyGPUArray(dWU);
  mxGPUDestroyGPUArray(UtU);
  mxGPUDestroyGPUArray(W);
  
}
